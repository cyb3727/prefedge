#include <limits.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#define BYTE_INDEX(n) ((n)/8)
#define BIT_INDEX(n) ((n) & 7)
typedef struct csr_st {
  unsigned long *index;     /* Mapped into VM -- do not access directly */
  unsigned long *index_aux; /* Mapped into VM -- do not access directly */
  unsigned char *calist_map; /* Mapped into VM -- do not access directly */
  unsigned char *aux;         /* Mapped into VM -- do not access directly */
  unsigned long vertex_cnt;
  unsigned long alist_entries;
  unsigned long calist_bytes;
  unsigned long aux_bytes;
  int fd_index;
  int fd_index_aux;
  int fd_calist;
  int fd_aux;
  unsigned long assume_undirected;
  unsigned long bits_per_edge;
} csr_t;

typedef struct csr_edge_iterator_st {
  unsigned long offset;        /* Offset for next edge             */
  unsigned long offset_stop;   /* Offset to stop                   */
  unsigned long neighbour;     /* Set by iterator                  */
  char incoming;               /* Set by iterator if in-edge       */
} csr_edge_iterator_t;


typedef struct bfs_metadata_st {
  int touched;
} bfs_metadata_t;


extern bfs_metadata_t *metadata;
__device__ unsigned long d_visited;
__constant__ int bits_per_edge;
__constant__ unsigned long vertex_cnt;
__constant__ unsigned long alist_bits;

__device__ static int d_csr_iter_step(unsigned char *calist_map, csr_edge_iterator_t *iter)
{
  if(iter->offset == iter->offset_stop) {
    return -1;
  }
  /* Works only for little endian ! */
  unsigned long tmp;
  unsigned long start_byte = BYTE_INDEX(iter->offset);
  unsigned long stop_byte = BYTE_INDEX(iter->offset + bits_per_edge - 1);
  memcpy(&tmp, &calist_map[start_byte], (stop_byte - start_byte) + 1);
  tmp >>= BIT_INDEX(iter->offset);
  tmp &= ((((unsigned long)1) << bits_per_edge) - 1);
  iter->incoming = ((tmp & 1) != 0);
  tmp >>= 1;
  iter->neighbour = (unsigned long)tmp;
  iter->offset += bits_per_edge;
  return 0;
}

__device__ static void d_csr_init_edge_iterator(unsigned long *index,
    unsigned long vertex,
    csr_edge_iterator_t *iter)
{


  iter->offset = index[vertex];
  if(vertex < (vertex_cnt - 1)) {
    iter->offset_stop = index[vertex + 1];
  }
  else {
    iter->offset_stop = alist_bits;
    //graph->alist_entries*graph->bits_per_edge;
  }
}


__global__ void kBFS(int* metadata, unsigned long* index, unsigned char* calist_map, int level) {

  unsigned long current_vertex = blockIdx.x*blockDim.x + threadIdx.x;
  int WARP_ID = current_vertex ;
  if ((WARP_ID < vertex_cnt) && (metadata[WARP_ID] == level)){
    csr_edge_iterator_t iter;
    d_csr_init_edge_iterator(index, WARP_ID, &iter);
    while (d_csr_iter_step(calist_map, &iter) == 0) {
//      if (!iter.incoming) {
	unsigned long target = iter.neighbour;
	/*	if (metadata[target]==0) {
		metadata[target] = level+1;
		d_visited = level+1;
		}
	 */	
	if (atomicCAS(&metadata[target], 0, level+1) == 0) {
	  atomicAdd((unsigned long long*)&d_visited,1ULL);
	}
  //    }
    }
  }
}


extern "C" unsigned long bfs(csr_t *graph, unsigned long start_node) {
  unsigned long* calist_index, *calist_index_cp;
  unsigned char* calist_map, *calist_map_cp;
  char *env_var;
  int TPB = 32;
  env_var = getenv("OMP_NUM_THREADS");
  if(env_var != NULL) {
    TPB = atol(env_var);
  }

  int gridBlocks = ((graph->vertex_cnt) / TPB) + 1;
  hipSetDeviceFlags(hipDeviceMapHost);
  hipHostAlloc(&calist_index_cp, graph->vertex_cnt * sizeof(unsigned long), hipHostMallocMapped);
  memcpy(calist_index_cp, graph->index, graph->vertex_cnt * sizeof(unsigned long));
  //cudaHostRegister(graph->index, graph->vertex_cnt * sizeof(unsigned long), cudaHostRegisterMapped);
  hipHostGetDevicePointer(&calist_index, calist_index_cp, 0);
  //cudaHostRegister(graph->calist_map, graph->calist_bytes, cudaHostRegisterMapped);
  hipHostAlloc(&calist_map_cp, graph->calist_bytes, hipHostMallocMapped);
  memcpy(calist_map_cp, graph->calist_map, graph->calist_bytes);
  hipHostGetDevicePointer(&calist_map, calist_map_cp, 0);
  //cudaHostAlloc(&metadata, graph->vertex_cnt * sizeof(bfs_metadata_t), cudaHostAllocMapped);

  int* d_metadata;
  hipMalloc(&d_metadata, graph->vertex_cnt * sizeof(bfs_metadata_t));
  hipMemset(d_metadata, 0, graph->vertex_cnt * sizeof(bfs_metadata_t));
  //cudaHostGetDevicePointer(&d_metadata, metadata, 0);
  //cudaMalloc((void**) &d_visited, sizeof(unsigned long));
  //cudaMalloc((void**) &bits_per_edge, sizeof(int));
  //cudaMalloc((void**) &vertex_cnt, sizeof(unsigned long));
  //cudaMalloc((void**) &alist_bits, sizeof(unsigned long));
  hipMemcpyToSymbol(HIP_SYMBOL(bits_per_edge), &(graph->bits_per_edge), sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(vertex_cnt), &(graph->vertex_cnt), sizeof(unsigned long));
  unsigned long halist_bits = graph->bits_per_edge * graph->alist_entries;
  hipMemcpyToSymbol(HIP_SYMBOL(alist_bits), &halist_bits, sizeof(unsigned long));
  hipDeviceSynchronize();
  unsigned long visited = 1;
  int level = 0;
  //metadata[start_node].touched = 1;
  hipMemcpy(&d_metadata[start_node], &visited, sizeof(bfs_metadata_t),hipMemcpyHostToDevice);
  while (1) {
    level++;
    printf("> Running iteration %d ...\n", level);
    hipMemcpyToSymbol(HIP_SYMBOL(d_visited), &visited, sizeof(visited));

    assert(hipGetLastError()==0);
    kBFS<<<gridBlocks,TPB>>>((int*) d_metadata, calist_index, calist_map, level);
    assert(hipGetLastError()==0);
    hipDeviceSynchronize();
    unsigned long old_visited = visited;
    hipMemcpyFromSymbol(&visited, HIP_SYMBOL(d_visited), sizeof(visited));
    if (old_visited == visited) break;
    printf("> Visited %ld ...\n", visited);

  }

  return 1;
}

